#include "hip/hip_runtime.h"
#include <stdio.h>
#include<time.h>
#define SIZE 10

__global__ void VectorMatrixMult(int a[], int b[], int c[], int n)
{
	int i = threadIdx.x;

	if(i < n){
		//j is used for iterating through columns;---
		for(int j=0; j<SIZE; j++){
			c[i] +=(a[j] * *(b + i*SIZE + j));
		}
	}
}

int main()
{
	int *a, *b, *c;
	clock_t t;

	//vector
	a = (int*)malloc(SIZE * sizeof(int));
	//matrix
	b = (int*)malloc(SIZE * SIZE * sizeof(int));
	//result
	c = (int*)malloc(SIZE * sizeof(int));

	for (int i = 0; i < SIZE; i++){
		a[i] = i+1;
		for (int j = 0; j < SIZE; j++){
			*(b + i*SIZE + j) = i*j;
			//int index=blockDim.x * blockIdx.x + threadIdx.x;
		}
	}

	int *d_a, *d_b, *d_c;
	int size=SIZE * sizeof(int);
	int size2d=SIZE * SIZE * sizeof(int);
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size2d);
	hipMalloc(&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size2d, hipMemcpyHostToDevice);
t=clock();
	VectorMatrixMult <<< 1, SIZE >>> (d_a, d_b, d_c, SIZE);

	hipDeviceSynchronize();
	t=clock()-t;
double t2=((double)t/CLOCk_PER_SEC);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	printf("Vector: \n");
	for (int i = 0; i < SIZE; i++){
		printf("%d ", a[i]);
	}
	printf("\n");
	printf("Matrix: \n");
	for (int i = 0; i < SIZE; i++){
		for (int j = 0; j < SIZE; j++){
			printf("%d ", *(b + i*SIZE + j));
		}
		printf("\n");
	}
	printf("Product: \n");
	for (int i = 0; i < SIZE; i++){
		printf("%d ", c[i]);
	}
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(a);
	free(b);
	free(c);

	return 0;
}
